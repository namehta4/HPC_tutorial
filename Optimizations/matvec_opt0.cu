#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 256

// CUDA kernel for matrix-vector multiplication and bias addition
__global__ void matvec_kernel(int n, int m,
		              const double* x,
			      const double* b,
			      const double* w,
			      double* a) {
    // Each thread computes one output row (dot product)
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= n) return;

    // Use local accumulator
    double sum = 0.0;

    // Loop calculating the matrix-vector dot product
    int j = 0;
    for (j = 0; j < m; j++) {
        sum += w[row * m + j] * x[j];
    }
    a[row] = sum + b[row];
}

int main() {
    const int n = 80000;  // rows
    const int m = 8000;   // columns
    const int nsteps = 100;

    // Allocate host memory
    double *x = (double*)malloc(m * sizeof(double));
    double *b = (double*)malloc(n * sizeof(double));
    double *a = (double*)malloc(n * sizeof(double));
    double *w = (double*)malloc((size_t)n * m * sizeof(double));

    // Initialize data
    for (int i = 0; i < m; ++i) x[i] = 2.0;
    for (int i = 0; i < n; ++i) {
        b[i] = 1.0;
        a[i] = 0.0;
        for (int j = 0; j < m; ++j)
            w[i * m + j] = 1.0;
    }

    // Device allocations
    double *d_x, *d_b, *d_w, *d_a;
    hipMalloc(&d_x, m * sizeof(double));
    hipMalloc(&d_b, n * sizeof(double));
    hipMalloc(&d_w, (size_t)n * m * sizeof(double));
    hipMalloc(&d_a, n * sizeof(double));

    // Copy data to device
    hipMemcpy(d_x, x, m * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_w, w, (size_t)n * m * sizeof(double), hipMemcpyHostToDevice);

    // Kernel configuration
    dim3 block(BLOCK_SIZE);
    dim3 grid((n + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Timing with CUDA events (GPU-only timing)
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    for (int i = 0; i < nsteps; ++i) {
        matvec_kernel<<<grid, block>>>(n, m, d_x, d_b, d_w, d_a);
    }
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms = 0.0f;
    hipEventElapsedTime(&ms, start, stop);
    printf("Time taken for %d steps: %.3f sec\n", nsteps, ms / 1000.0f);

    // Copy result back
    hipMemcpy(a, d_a, n * sizeof(double), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_x);
    hipFree(d_b);
    hipFree(d_w);
    hipFree(d_a);
    free(x);
    free(b);
    free(a);
    free(w);

    return 0;
}

